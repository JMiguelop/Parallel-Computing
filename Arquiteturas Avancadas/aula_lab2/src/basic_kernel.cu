/**************************************************************
 * 
 * --== Simple CUDA kernel ==--
 * author: ampereira
 * 
 *
 * Fill the rest of the code through the following steps:
 * -> allocate the device memory
 * -> copy the inputs to the device
 * -> call the kernel
 * -> copy the output to the host
 * 
 * extra points for
 * -> reversing the output array on the device
 *
 **************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK

using namespace std;

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

// Fill the input parameters and kernel qualifier
__global__ void vecAdditionKernel (float *a, float *b, float *c) {
	//for(int i = 0; i < SIZE; i++) {
	//	c[i] = a[i] + b[i];
	//}
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	c[tid] = a[tid] + b[tid];
}

int main( int argc, char** argv) {
	// declare variable with size of the array in bytes
	int bytes = SIZE*sizeof(float);

	// arrays on the host
	float a[SIZE], b[SIZE], c[SIZE];

	// pointers to the device memory
	float *dev_a, *dev_b, *dev_c;

	// fills the arrays
	for (unsigned i = 0; i < SIZE; ++i) {
		a[i] = (float) rand() / RAND_MAX;
		b[i] = (float) rand() / RAND_MAX;
	}

	// allocate the memory on the device
	hipMalloc((void**)&dev_a, bytes);
	hipMalloc((void**)&dev_b, bytes);
	hipMalloc((void**)&dev_c, bytes);

	checkCUDAError("mem allocation");

	

	// copy inputs to the device
	hipMemcpy(dev_a, &a, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, sizeof(float), hipMemcpyHostToDevice);	
	
	checkCUDAError("memcpy h->d");

	// launch the kernel
	dim3 dimGrid(NUM_BLOCKS);
	dim3 dimBlock(NUM_THREADS_PER_BLOCK);
	vecAdditionKernel <<< dimBlock, dimGrid >>> (dev_a, dev_b, dev_c);

	checkCUDAError("kernel invocation");

	// copy the output to the host
	hipMemcpy(&c, dev_c, bytes, hipMemcpyDeviceToHost);	
	
	// free the device memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	checkCUDAError("mem free");

	//Print result just for feedback
	std::cout << "finished adding matrix\n";

	return 0;
}
